#include "gtest/gtest.h"
#include "dspapple/cuda/utils.cuh"

TEST(CUDA, Conversion)
{
    size_t SAMPLES = 1024;
    int16_t* input;
    hipfftComplex* output;

    hipMallocManaged(&input, SAMPLES * sizeof(int16_t));
    ASSERT_EQ(hipGetLastError(), hipSuccess) << hipGetErrorString(hipGetLastError());

    for(size_t i=0; i < SAMPLES; ++i)
    {
        input[i] = i;
    }

    hipMallocManaged(&output, (SAMPLES / 2) * sizeof(hipfftComplex));
    ASSERT_EQ(hipGetLastError(), hipSuccess) << hipGetErrorString(hipGetLastError());

    dspapple::ConvertVecGpu(input, output, SAMPLES);
    hipDeviceSynchronize();

    const float EPS = 1e-5;

    for(size_t i=0; i < SAMPLES; i += 2)
    {
        EXPECT_NEAR(output[i / 2].x, i / 32767.0f, EPS);
        EXPECT_NEAR(output[i / 2].y, (i + 1) / 32767.0f, EPS);
    }

    hipFree(input);
    hipFree(output);
}
