#include "gtest/gtest.h"
#include "dspapple/cuda/utils.cuh"

TEST(CUDA, Conversion)
{
    size_t SAMPLES = 1024;
    int16_t* input;
    hipfftComplex* output;

    hipMallocManaged(&input, SAMPLES * sizeof(int16_t));
    ASSERT_EQ(hipGetLastError(), hipSuccess) << hipGetErrorString(hipGetLastError());

    for(size_t i=0; i < SAMPLES; ++i)
    {
        input[i] = i;
    }

    hipMallocManaged(&output, (SAMPLES / 2) * sizeof(hipfftComplex));
    ASSERT_EQ(hipGetLastError(), hipSuccess) << hipGetErrorString(hipGetLastError());

    dspapple::ConvertVecGpu(input, output, SAMPLES);
    hipDeviceSynchronize();

    for(size_t i=0; i < SAMPLES; i += 2)
    {
        EXPECT_EQ(output[i / 2].x, i);
        EXPECT_EQ(output[i / 2].y, i + 1);
    }

    hipFree(input);
    hipFree(output);
}
